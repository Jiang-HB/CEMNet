#include "hip/hip_runtime.h"
#include "../cuda_utils.h"
#include "ops_cuda_kernel.h"
#include "math.h"

// input: srcs(B, 3, n) tgt(3, n)
// output: out(b, n)
__global__ void closest_point_cuda_kernel(int b, int c, int n, float *closest_points, const float *srcs, const float *tgt)
{
    int idx_block = blockIdx.x;
    int idx_thread = threadIdx.x;

    if (idx_block < b && idx_thread < n)
    {
        float min_val = 1000.0;
        int idx = -1;
        for (int k = 0; k < n; k += 1)
        {
            float val = 0.0;
            for (int i = 0; i < c; i += 1)
            {
                val += (srcs[idx_block * c * n + i * n + idx_thread] - tgt[i * n + k]) * (srcs[idx_block * c * n + i * n + idx_thread] - tgt[i * n + k]);
            }
            if (val < min_val)
            {
                min_val = val;
                idx = k;
            }
        }
        closest_points[idx_block * n * c + idx_thread] = tgt[idx];
        closest_points[idx_block * n * c + n + idx_thread] = tgt[n + idx];
        closest_points[idx_block * n * c + 2 * n + idx_thread] = tgt[2 * n + idx];
    }
}

void closest_point_cuda_launcher(int b, int c, int n, float *closest_points, const float *srcs, const float *tgt)
{
    dim3 grid(b, 1, 1);
    dim3 block(n, 1, 1);
    closest_point_cuda_kernel<<<grid, block, 0>>>(b, c, n, closest_points, srcs, tgt);
}

// input: srcs(b, 3, n) tgt(3, n) distances(b)
// output: None
__global__ void mc_distance_cuda_kernel(int b, int c, int n, float r, const float *srcs, const float *tgt, float * distances, int *min_idxs)
{
    int idx_block = blockIdx.x;   // b_idx
    int idx_thread = threadIdx.x; // n_idx
    float rr = r * r;
    if (idx_block < b && idx_thread < n)
    {
        float min_distance1 = 1000.0;
        float min_distance2 = 1000.0;
        int min_idx1 = -1;
        int min_idx2 = -1;
        for (int i = 0; i < n; i += 1)
        {
            float distance1 = 0.0;
            float distance2 = 0.0;
            for (int j = 0; j < c; j += 1)
            {
                distance1 += (srcs[idx_block * c * n + j * n + idx_thread] - tgt[j * n + i]) * (srcs[idx_block * c * n + j * n + idx_thread] - tgt[j * n + i]);
                distance2 += (srcs[idx_block * c * n + j * n + i] - tgt[j * n + idx_thread]) * (srcs[idx_block * c * n + j * n + i] - tgt[j * n + idx_thread]);
            }
            if (distance1 < min_distance1)
            {
                min_distance1 = distance1;
                min_idx1 = i;
            }
            if (distance2 < min_distance2)
            {
                min_distance2 = distance2;
                min_idx2 = i;
            }
        }
        if (min_distance1 <= rr)
        {
            distances[idx_block * n * 2 + idx_thread * 2] = (1.0 - sqrt((float)(min_distance1)) / r);
            min_idxs[idx_block * n * 2 + idx_thread * 2] = min_idx1;
        }
        if (min_distance2 <= rr)
        {
            distances[idx_block * n * 2 + idx_thread * 2 + 1] = (1.0 - sqrt((float)(min_distance2)) / r);
            min_idxs[idx_block * n * 2 + idx_thread * 2 + 1] = min_idx2;
        }
    }
}

void mc_distance_cuda_launcher(int b, int c, int n, float r, const float *srcs, const float *tgt, float * distances, int *min_idxs)
{
    dim3 grid(b, 1, 1);
    dim3 block(n, 1, 1);
    mc_distance_cuda_kernel<<<grid, block, 0>>>(b, c, n, r, srcs, tgt, distances, min_idxs);
}

// input: srcs(b, 3, n) tgt(3, n) distances(b)
// output: None
__global__ void cd_distance_cuda_kernel(int b, int c, int n, const float *srcs, const float *tgt, float * distances)
{
    int idx_block = blockIdx.x;   // b_idx
    int idx_thread = threadIdx.x; // n_idx

    if (idx_block < b && idx_thread < n)
    {
        float min_distance1 = 1000.0;
        float min_distance2 = 1000.0;
        for (int i = 0; i < n; i += 1)
        {
            float distance1 = 0.0;
            float distance2 = 0.0;
            for (int j = 0; j < c; j += 1)
            {
                distance1 += (srcs[idx_block * c * n + j * n + idx_thread] - tgt[j * n + i]) * (srcs[idx_block * c * n + j * n + idx_thread] - tgt[j * n + i]);
                distance2 += (srcs[idx_block * c * n + j * n + i] - tgt[j * n + idx_thread]) * (srcs[idx_block * c * n + j * n + i] - tgt[j * n + idx_thread]);
            }
            if (distance1 < min_distance1)
            {
                min_distance1 = distance1;
            }
            if (distance2 < min_distance2)
            {
                min_distance2 = distance2;
            }
        }
        distances[idx_block * n * 2 + idx_thread * 2] = min_distance1;
        distances[idx_block * n * 2 + idx_thread * 2 + 1] = min_distance2;
    }
}

void cd_distance_cuda_launcher(int b, int c, int n, const float *srcs, const float *tgt, float * distances)
{
    dim3 grid(b, 1, 1);
    dim3 block(n, 1, 1);
    cd_distance_cuda_kernel<<<grid, block, 0>>>(b, c, n, srcs, tgt, distances);
}


// input: srcs(b, 3, n) tgt(3, n) distances(b)
// output: None
__global__ void cycle_distance_cuda_kernel(int b, int c, int n, int N, const float *srcs, const float *tgt, float *distances, int *min_idxs, float * n_distances, int * n_idxs)
{
    int idx_block = blockIdx.x;   // b_idx
    int idx_thread = threadIdx.x; // n_idx
    if (idx_block < b && idx_thread < n)
    {

        for (int i = 0; i < N; i += 1)
        {
           n_distances[i] = 100.0;
           n_idxs[i] = -1;
        }

        // cal xy_idxs and xy_distance
        float min_distance1 = 1000.0;
        int min_idx1 = -1;
        for (int i = 0; i < n; i += 1)
        {
            float distance1 = 0.0;
            for (int j = 0; j < c; j += 1)
            {
                distance1 += (srcs[idx_block * c * n + j * n + idx_thread] - tgt[j * n + i]) * (srcs[idx_block * c * n + j * n + idx_thread] - tgt[j * n + i]);
            }
            if (distance1 < min_distance1)
            {
                min_distance1 = distance1;
                min_idx1 = i;
            }
        }

        // cal yx_n_idxs and yx_n_distance
        int cnt = 0;
//        for (int i = 0; i < n; i += 1)
//        {
//            if (cnt >= N)
//            {
//                break;
//            }
//            float distance2 = 0.0;
//            for (int j = 0; j < c; j += 1)
//            {
//                distance2 += (srcs[idx_block * c * n + j * n + i] - tgt[j * n + min_idx1]) * (srcs[idx_block * c * n + j * n + i] - tgt[j * n + min_idx1]);
//            }
//            if (distance2 < distance1)
//            {
//                cnt += 1;
//            }
//        }

        if (cnt < N)
        {
            distances[idx_block * n + idx_thread] = - min_distance1;
            min_idxs[idx_block * n + idx_thread] = min_idx1;
        }
    }
}

void cycle_distance_cuda_launcher(int b, int c, int n, int N, const float *srcs, const float *tgt, float * distances, int *min_idxs, float * n_distances, int * n_idxs)
{
    dim3 grid(b, 1, 1);
    dim3 block(n, 1, 1);
    cycle_distance_cuda_kernel<<<grid, block, 0>>>(b, c, n, N, srcs, tgt, distances, min_idxs, n_distances, n_idxs);
}